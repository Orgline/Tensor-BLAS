#include "../include/TensorBLAS.h" 

void tc_cumpsgemm_trsm_p2(hipblasHandle_t handle, cumpsgemm::handle_t cumpsgemm_handle, long int m, long int n, float* A, long int lda, float* B, long int ldb, long int nb)
{
    if(n <= nb)
    {
        //startTimer();
        hipblasStrsm(handle,
            HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
            HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
            m, n, &sone,
            A, lda,
            B, ldb
        );
        return;
    }
    
    tc_cumpsgemm_trsm_p2(handle, cumpsgemm_handle, m, n/2, A, lda, B, ldb, nb);

    cumpsgemm::gemm(
        cumpsgemm_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        m, n/2, n/2,
        &snegone,
        B, ldb,
        A+n/2, lda,
        &sone,
        B+n/2*ldb, ldb,
        CUMPSGEMM_FP16TCEC
        );

    tc_cumpsgemm_trsm_p2(handle, cumpsgemm_handle, m, n/2, A+n/2*lda+n/2, lda, B+n/2*ldb, ldb, nb);
}
 
void tc_cumpsgemm_trsm_p3(hipblasHandle_t handle, cumpsgemm::handle_t cumpsgemm_handle,long int m, long int n, float* A, long int lda, float* B, long int ldb, long int nb)
{
    int length;
    int64_t* matSize = find_mat_size_trsm(n, &length);
    long int offset;
    long int rest_n = n;

    for(int i = length; i>=0; i--)
    {
        int64_t nn = matSize[i];
        if(i < length)
            offset += matSize[i + 1];
        else
            offset = 0;
        if(nn % 2048 == 0)
        {
            // printf("now nn=%d i = %d check ok\n", nn, i);
            tc_cumpsgemm_trsm_p2(handle, cumpsgemm_handle, m, nn, A+offset+offset*lda, lda, B+offset*ldb, ldb, nb);
            // printf("check ok\n");
        }
        else
        {
            
            hipblasStrsm(handle,
                HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
                HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
                m, nn, &sone,
                A+offset+offset*lda, lda,
                B+offset*ldb, ldb
            );
            
        }
        
        if(i != 0)
        {
            rest_n -=  nn;
            cumpsgemm::gemm(
                    cumpsgemm_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_T,
                    m, rest_n, nn,
                    &snegone,
                    B+offset*ldb, ldb,
                    A+offset+nn+offset*lda, lda,
                    &sone,
                    B+(offset+nn)*ldb, ldb,
                    CUMPSGEMM_FP16TCEC
                    );

        }
        

    }

}

void tc_cumpsgemm_trsm(hipblasHandle_t handle, cumpsgemm::handle_t cumpsgemm_handle,long int m, long int n, float* A, long int lda, float* B, long int ldb, long int nb)
{
    if(n%2||m%2) {
        float *A_, *B_;
        long int N = n, M = m, lda_, ldb_;
        n += n%2;
        m += m%2;
        lda_ = lda + lda%2;
        ldb_ = ldb + ldb%2;
        hipMalloc(&A_, sizeof(float)*n*n);
        hipMalloc(&B_, sizeof(float)*m*n);
        printf("%ld, %ld\n", m, n);
        dim3 grid1((n+31)/32, (n+31)/32);
        dim3 block(32,32);
        setInitialValue<<<grid1, block>>>(n, n ,A_, lda_, 0.0);
        dim3 grid2((m+31)/32, (n+31)/32);
        setInitialValue<<<grid2, block>>>(m, n ,B_, ldb_, 0.0);

        dim3 grid3((N+31)/32, (N+31)/32);
        matrixCpy<<<grid3, block>>>(N, N, A, lda, A_, lda_);
        dim3 grid4((M+31)/32, (N+31)/32);
        matrixCpy<<<grid4, block>>>(M, N, B, ldb, B_, ldb_);

        tc_cumpsgemm_trsm_p3(handle, cumpsgemm_handle, m, n, A_, lda_, B_, ldb_, nb);

        matrixCpy<<<grid4, block>>>(M, N, B_, ldb_, B, ldb);
        printf("check ok\n");
        hipFree(A_);
        hipFree(B_);

    }
    else {
        tc_cumpsgemm_trsm_p3(handle, cumpsgemm_handle, m, n, A, lda, B, ldb, nb);
    }
}