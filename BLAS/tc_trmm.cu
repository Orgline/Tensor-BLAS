#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h" 


void tc_trmm_p2(hipblasHandle_t handle, long int m, long int n, float alpha, __half* Ah, long int lda, __half* Bh, long int ldb, float beta, float* C, long int ldc, long int nb)
{
    //beginTimer();
    hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                nb, n, nb, &alpha,
                                Ah, HIP_R_16F, lda, nb+nb*lda,
                                Bh, HIP_R_16F, ldb, nb,
                                &beta, C, HIP_R_32F, ldc, nb,
                                m/nb, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    //float ms = endTimer();
    //printf("batch gemm size %dx%dx%d takes %fms, rate is %f TFLOPs\n", nb, n, nb, ms, 2.0*nb*n*nb*m/nb/ms/1e9);
    for(long int i = 1; m / nb / i / 2 >= 1; i*=2)
    {
        //beginTimer();
        hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                   i*nb, n, i*nb, &alpha,
                                   Ah+i*nb, HIP_R_16F, lda, 2*(i*nb+i*nb*lda),
                                   Bh, HIP_R_16F, ldb, 2*i*nb,
                                   &sone, C+i*nb, HIP_R_32F, ldc, 2*i*nb,
                                   m/nb/i/2, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        //ms = endTimer();
        //printf("batch gemm size %dx%dx%d takes %fms, rate is %f TFLOPs\n", i*nb, n, i*nb, ms, 2.0*i*nb*n*i*nb*m/nb/i/2.0/ms/1e9);
    }
}

void tc_trmm(hipblasHandle_t handle, long int m, long int n, float alpha, float* A, long int lda, float* B, long int ldb, float* C, long int ldc, __half* hwork, long int nb)
{
    int length;
    int64_t* matSize = find_mat_size_syrk(m, &length);
    // for(int i = 0; i<=length; i++)
    // {
    //     printf("%ld ", matSize[i]);
    // }
    // printf("\n");
    int offset = 0;
    int rest_m = m;

    __half* Ah = hwork;
    __half* Bh = hwork + m*m;
    dim3 gridA((m+31)/32, (m+31)/32);
    dim3 block(32,32);
    s2h<<<gridA, block>>>(m, m, A, lda, Ah, lda);

    dim3 gridB((m+31)/32, (n+31)/32);
    s2h<<<gridB, block>>>(m, n, B, ldb, Bh, ldb);


    for(int i = length; i>=0; i--)
    {
        int mm = matSize[i];
        float beta;
        if(i < length)
            offset += matSize[i + 1];
        else
            offset = 0;
        if(i == length)
            beta = szero;
        else
            beta = sone;
        if (mm % 8192 == 0)
        {
            tc_trmm_p2(handle, mm, n, alpha, Ah+offset+offset*lda, lda, Bh+offset, ldb, beta, C+offset, ldc, nb);
        }
        else
        {
            //printf("offset = %d\n", offset);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, mm, n, mm,
                &alpha, Ah+offset+offset*lda, HIP_R_16F, lda, Bh+offset, HIP_R_16F, ldb,
                &beta, C+offset, HIP_R_32F, ldc, HIP_R_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        if(i != 0)
        {
            rest_m -= mm;         
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rest_m, n, mm,
                &alpha, Ah+offset+mm+offset*lda, HIP_R_16F, lda, Bh+offset, HIP_R_16F, ldb,
                &beta, C+offset+mm, HIP_R_32F, ldc, HIP_R_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
    }
}