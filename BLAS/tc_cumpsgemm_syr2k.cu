#include "../include/TensorBLAS.h" 

void tc_cumpsgemm_syr2k_p2(cumpsgemm::handle_t cumpsgemm_handle, long int n, long int k, float alpha, float* A, long int lda, float* B, long int ldb, float beta, float* C, long int ldc, long int nb)
{
    cumpsgemm::gemm_stridedBatch<float>(
				cumpsgemm_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_T,
				nb, nb, k,
				&alpha,
				A, lda, nb,
				B, ldb, nb,
				&beta,
				C, ldc, nb+nb*lda,
				n/nb,
				CUMPSGEMM_AUTO
				);
    cumpsgemm::gemm_stridedBatch<float>(
				cumpsgemm_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_T,
				nb, nb, k,
				&alpha,
				B, ldb, nb,
				A, lda, nb,
				&sone,
				C, ldc, nb+nb*lda,
				n/nb,
				CUMPSGEMM_AUTO
				);

    for(int i = 1;n / nb / i / 2 >= 1; i*=2)
    {
        cumpsgemm::gemm_stridedBatch<float>(
				cumpsgemm_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_T,
				i*nb, i*nb, k,
				&alpha,
				A+i*nb, lda, 2*i*nb,
				B, ldb, 2*i*nb,
				&beta,
				C+i*nb, ldc, 2*(i*nb+i*nb*lda),
				n/nb/i/2,
				CUMPSGEMM_AUTO
				);
        cumpsgemm::gemm_stridedBatch<float>(
				cumpsgemm_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_T,
				i*nb, i*nb, k,
				&alpha,
				B+i*nb, ldb, 2*i*nb,
				A, lda, 2*i*nb,
				&sone,
				C+i*nb, ldc, 2*(i*nb+i*nb*lda),
				n/nb/i/2,
				CUMPSGEMM_AUTO
				);
    }
}

void tc_cumpsgemm_syr2k(cumpsgemm::handle_t cumpsgemm_handle, long int n, long int k,  float alpha, float* A, long int lda, float* B, long int ldb, float beta, float* C, long int ldc, long int nb)
{
    
    int length;
    int64_t* matSize = find_mat_size_syrk(n, &length);
    int offset;
    int rest_n = n;

    for(int i = length; i>=0; i--)
    {

        int nn = matSize[i];
        
        if(i < length)
            offset += matSize[i + 1];
        else
            offset = 0;

        if(nn % 8192 ==0 )
        {
            tc_cumpsgemm_syr2k_p2(cumpsgemm_handle, nn, k, alpha, A+offset, lda, B+offset, ldb, beta, C+offset+offset*ldc, ldc, nb);
        }
        else
        {
            cumpsgemm::gemm(
                    cumpsgemm_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_T,
                    nn, nn, k,
                    &alpha,
                    A+offset, lda,
                    B+offset, ldb,
                    &beta,
                    C+offset+offset*ldc, ldc,
                    CUMPSGEMM_AUTO
                    );
            cumpsgemm::gemm(
                    cumpsgemm_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_T,
                    nn, nn, k,
                    &alpha,
                    B+offset, ldb,
                    A+offset, lda,
                    &sone,
                    C+offset+offset*ldc, ldc,
                    CUMPSGEMM_AUTO
                    );
        }
        if(i != 0)
        {
            rest_n -=  nn;
            cumpsgemm::gemm(
                    cumpsgemm_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_T,
                    rest_n, nn, k,
                    &alpha,
                    A+offset+nn, lda,
                    B+offset, ldb,
                    &beta,
                    C+offset+offset*ldc+nn, ldc,
                    CUMPSGEMM_AUTO
                    );
            cumpsgemm::gemm(
                    cumpsgemm_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_T,
                    rest_n, nn, k,
                    &alpha,
                    B+offset+nn, ldb,
                    A+offset, lda,
                    &sone,
                    C+offset+offset*ldc+nn, ldc,
                    CUMPSGEMM_AUTO
                    );
        }
        else
            return;
        
    }
    return;
}