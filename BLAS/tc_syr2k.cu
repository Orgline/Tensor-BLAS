#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h" 

void tc_syr2k_p2(hipblasHandle_t handle, long int n, long int k, float alpha, __half* Ah, long int lda, __half* Bh, long int ldb, float beta, float* C, long int ldc, long int nb)
{
    //printf("tc_syrk_p2\n");
    hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                nb, nb, k, &alpha,
                                Ah, HIP_R_16F, lda, nb,
                                Bh, HIP_R_16F, ldb, nb,
                                &beta, C, HIP_R_32F, ldc, nb+nb*lda,
                                n/nb, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                nb, nb, k, &alpha,
                                Bh, HIP_R_16F, ldb, nb,
                                Ah, HIP_R_16F, lda, nb,
                                &sone, C, HIP_R_32F, ldc, nb+nb*lda,
                                n/nb, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    for(int i = 1;n / nb / i / 2 >= 1; i*=2)
    {
        hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                   i*nb, i*nb, k, &alpha,
                                   Ah+i*nb, HIP_R_16F, lda, 2*i*nb,
                                   Bh, HIP_R_16F, ldb, 2*i*nb,
                                   &beta, C+i*nb, HIP_R_32F, ldc, 2*(i*nb+i*nb*lda),
                                   n/nb/i/2, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                   i*nb, i*nb, k, &alpha,
                                   Bh+i*nb, HIP_R_16F, ldb, 2*i*nb,
                                   Ah, HIP_R_16F, lda, 2*i*nb,
                                   &sone, C+i*nb, HIP_R_32F, ldc, 2*(i*nb+i*nb*lda),
                                   n/nb/i/2, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
}

void tc_syr2k_p3(hipblasHandle_t handle, long int n, long int k,  float alpha, float* A, long int lda, float* B, long int ldb, float beta, float* C, long int ldc, __half* hwork, long int nb)
{
    
    int length;
    int64_t* matSize = find_mat_size_syrk(n, &length);
    // for(int i = 0; i<=length; i++)
    // {
    //     printf("%ld ", matSize[i]);
    // }
    // printf("\n");
    int offset;
    int rest_n = n;

    __half *Ah = hwork;
    __half *Bh = hwork + n*k;

    dim3 grid((n+31)/32, (k+31)/32);
    dim3 block(32,32);
    s2h<<<grid, block>>>(n, k, A, lda, Ah, lda);
    s2h<<<grid, block>>>(n, k, B, ldb, Bh, ldb);

    for(int i = length; i>=0; i--)
    {

        int nn = matSize[i];
        

        if(i < length)
            offset += matSize[i + 1];
        else
            offset = 0;

        if(nn % 8192 ==0 )
        {
            tc_syr2k_p2(handle, nn, k, alpha, Ah+offset, lda, Bh+offset, ldb, beta, C+offset+offset*ldc, ldc, nb);
        }
        else
        {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nn, nn, k,
                    &alpha, Ah+offset, HIP_R_16F, lda, Bh+offset, HIP_R_16F, ldb,
                    &beta, C+offset+offset*ldc, HIP_R_32F, ldc, HIP_R_32F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nn, nn, k,
                    &alpha, Bh+offset, HIP_R_16F, ldb, Ah+offset, HIP_R_16F, lda,
                    &sone, C+offset+offset*ldc, HIP_R_32F, ldc, HIP_R_32F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        if(i != 0)
        {
            rest_n -=  nn;
            //printf("rest_n = %d, nn = %d, offset = %d\n", rest_n, nn, offset);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, rest_n, nn, k,
                    &alpha, Ah+offset+nn, HIP_R_16F, lda, Bh+offset, HIP_R_16F, ldb,
                    &beta, C+offset+offset*ldc+nn, HIP_R_32F, ldc, HIP_R_32F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, rest_n, nn, k,
                    &alpha, Bh+offset+nn, HIP_R_16F, ldb, Ah+offset, HIP_R_16F, lda,
                    &sone, C+offset+offset*ldc+nn, HIP_R_32F, ldc, HIP_R_32F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        else
            return;
        
    }
    return;
}
void tc_syr2k(hipblasHandle_t handle, long int n, long int k,  float alpha, float* A, long int lda, float* B, long int ldb, float beta, float* C, long int ldc, __half* hwork, long int nb)
{
    if(n%2||k%2) {
        float *A_, *C_, *B_;
        long int N = n, K = k, lda_, ldb_, ldc_;
        n += n%2;
        k += k%2;
        lda_ = lda + lda%2;
        ldb_ = ldb + ldb%2;
        ldc_ = ldc + ldc%2;
        hipMalloc(&A_, sizeof(float)*n*k);
        hipMalloc(&B_, sizeof(float)*n*k);
        hipMalloc(&C_, sizeof(float)*n*n);
        printf("%ld, %ld\n", n, k);
        dim3 grid((n+31)/32, (k+31)/32);
        dim3 block(32,32);
        setInitialValue<<<grid, block>>>(n, k ,A_, lda_, 0.0);
        setInitialValue<<<grid, block>>>(n, k ,B_, ldb_, 0.0);
        setInitialValue<<<grid, block>>>(n, n ,C_, ldc_, 1.0);

        matrixCpy<<<grid, block>>>(N, K, A, lda, A_, lda_);
        matrixCpy<<<grid, block>>>(N, K, B, ldb, B_, ldb_);


        tc_syr2k_p3(handle, n, k, alpha, A_, lda_, B_, ldb_, beta, C_, ldc_, hwork, nb);

        matrixCpy<<<grid, block>>>(N, N, C_, ldc_, C, ldc);

        printf("check ok\n");
        hipFree(A_);
        hipFree(B_);
        hipFree(C_);
    }
    else {
        tc_syr2k_p3(handle, n, k, alpha, A, lda, B, ldb, beta, C, ldc, hwork, nb);
    }
}