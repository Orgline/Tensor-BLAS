#include "../include/TensorBLAS.h" 

void tc_cumpsgemm_syrk_p2(cumpsgemm::handle_t cumpsgemm_handle, long int n, long int k, float alpha, float* A, long int lda, float beta, float* C, long int ldc, long int nb)
{
    cumpsgemm::gemm_stridedBatch<float>(
				cumpsgemm_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_T,
				nb, nb, k,
				&alpha,
				A, lda, nb,
				A, lda, nb,
				&beta,
				C, ldc, nb+nb*lda,
				n/nb,
				CUMPSGEMM_FP16TCEC
				);

    for(int i = 1;n / nb / i / 2 >= 1; i*=2)
    {
        cumpsgemm::gemm_stridedBatch<float>(
				cumpsgemm_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_T,
				i*nb, i*nb, k,
				&alpha,
				A+i*nb, lda, 2*i*nb,
				A, lda, 2*i*nb,
				&beta,
				C+i*nb, ldc, 2*(i*nb+i*nb*lda),
				n/nb/i/2,
				CUMPSGEMM_FP16TCEC
				);
    }
}

void tc_cumpsgemm_syrk_p3(cumpsgemm::handle_t cumpsgemm_handle, long int n, long int k,  float alpha, float* A, long int lda, float beta, float* C, long int ldc, long int nb) {
    int length;
    int64_t* matSize = find_mat_size_syrk(n, &length);
    int offset;
    int rest_n = n;

    for(int i = length; i>=0; i--)
    {
        int nn = matSize[i];
        if(i < length)
            offset += matSize[i + 1];
        else
            offset = 0; 
        if(nn % 8192 ==0 )
        {
            tc_cumpsgemm_syrk_p2(cumpsgemm_handle, nn, k, alpha, A+offset, lda, beta, C+offset+offset*ldc, ldc, nb);
        }
        else
        {
            cumpsgemm::gemm(
                    cumpsgemm_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_T,
                    nn, nn, k,
                    &alpha,
                    A+offset, lda,
                    A+offset, lda,
                    &beta,
                    C+offset+offset*ldc, ldc,
                    CUMPSGEMM_FP16TCEC
                    );
        }
        if(i != 0)
        {
            rest_n -=  nn;
            cumpsgemm::gemm(
                    cumpsgemm_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_T,
                    rest_n, nn, k,
                    &alpha,
                    A+offset+nn, lda,
                    A+offset, lda,
                    &beta,
                    C+offset+offset*ldc+nn, ldc,
                    CUMPSGEMM_FP16TCEC
                    );
        }
        else
            return;
        
    }

}

void tc_cumpsgemm_syrk(cumpsgemm::handle_t cumpsgemm_handle, long int n, long int k,  float alpha, float* A, long int lda, float beta, float* C, long int ldc, long int nb)
{
    if(n%2||k%2) {
        float *A_, *C_;
        long int N = n, K = k, lda_, ldc_;
        n += n%2;
        k += k%2;
        lda_ = lda + lda%2;
        ldc_ = ldc + ldc%2;
        hipMalloc(&A_, sizeof(float)*n*k);
        hipMalloc(&C_, sizeof(float)*n*n);
        printf("%ld, %ld\n", n, k);
        dim3 grid((k+31)/32, (n+31)/32);
        dim3 block(32,32);
        setInitialValue<<<grid, block>>>(n, n ,C_, ldc_, 1.0);
        setInitialValue<<<grid, block>>>(n, k ,A_, lda_, 0.0);
        matrixCpy<<<grid, block>>>(N, K, A, lda, A_, lda_);//lda lda_
        // printMatrixDeviceBlock("A.csv", N, K, A, lda);
        // printMatrixDeviceBlock("A_.csv", n, k, A_, lda_);

        tc_cumpsgemm_syrk_p3(cumpsgemm_handle, n, k, alpha, A_, lda_, beta, C_, ldc_, nb);

        matrixCpy<<<grid, block>>>(N, N, C_, ldc_, C, ldc);
        // printMatrixDeviceBlock("C.csv", N, K, C, N);
        // printMatrixDeviceBlock("C_.csv", n, k, C_, n);
        printf("check ok\n");
        hipFree(A_);
        hipFree(C_);
    }
    else {
        tc_cumpsgemm_syrk_p3(cumpsgemm_handle, n, k, alpha, A, lda, beta, C, ldc, nb);
    }
    return;
}