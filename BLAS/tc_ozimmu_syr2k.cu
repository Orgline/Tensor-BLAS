#include "../include/TensorBLAS.h" 

void tc_ozimmu_syr2k_p2(hipblasHandle_t handle, long int n, long int k, double alpha, double* A, long int lda, double* B, long int ldb, double beta, double* C, long int ldc, long int nb)
{
    hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                nb, nb, k, &alpha,
                                A, HIP_R_64F, lda, nb,
                                B, HIP_R_64F, ldb, nb,
                                &beta, C, HIP_R_64F, ldc, nb+nb*lda,
                                n/nb, HIP_R_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                nb, nb, k, &alpha,
                                B, HIP_R_64F, ldb, nb,
                                A, HIP_R_64F, lda, nb,
                                &sone, C, HIP_R_64F, ldc, nb+nb*lda,
                                n/nb, HIP_R_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    for(int i = 1;n / nb / i / 2 >= 1; i*=2)
    {
        hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                   i*nb, i*nb, k, &alpha,
                                   A+i*nb, HIP_R_64F, lda, 2*i*nb,
                                   B, HIP_R_64F, ldb, 2*i*nb,
                                   &beta, C+i*nb, HIP_R_64F, ldc, 2*(i*nb+i*nb*lda),
                                   n/nb/i/2, HIP_R_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                   i*nb, i*nb, k, &alpha,
                                   B+i*nb, HIP_R_64F, ldb, 2*i*nb,
                                   A, HIP_R_64F, lda, 2*i*nb,
                                   &sone, C+i*nb, HIP_R_64F, ldc, 2*(i*nb+i*nb*lda),
                                   n/nb/i/2, HIP_R_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
}

void tc_ozimmu_syr2k_p3(hipblasHandle_t handle, long int n, long int k,  double alpha, double* A, long int lda, double* B, long int ldb, double beta, double* C, long int ldc, long int nb)
{
    
    int length;
    int64_t* matSize = find_mat_size_syrk(n, &length);
    int offset;
    int rest_n = n;

    for(int i = length; i>=0; i--)
    {

        int nn = matSize[i];
        
        if(i < length)
            offset += matSize[i + 1];
        else
            offset = 0;

        if(nn % 8192 ==0 )
        {
            tc_ozimmu_syr2k_p2(handle, nn, k, alpha, A+offset, lda, B+offset, ldb, beta, C+offset+offset*ldc, ldc, nb);
        }
        else
        {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nn, nn, k,
                    &alpha, A+offset, HIP_R_64F, lda, B+offset, HIP_R_64F, ldb,
                    &beta, C+offset+offset*ldc, HIP_R_64F, ldc, HIP_R_64F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nn, nn, k,
                    &alpha, B+offset, HIP_R_64F, ldb, A+offset, HIP_R_64F, lda,
                    &sone, C+offset+offset*ldc, HIP_R_64F, ldc, HIP_R_64F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        if(i != 0)
        {
            rest_n -=  nn;
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, rest_n, nn, k,
                    &alpha, A+offset+nn, HIP_R_64F, lda, B+offset, HIP_R_64F, ldb,
                    &beta, C+offset+offset*ldc+nn, HIP_R_64F, ldc, HIP_R_64F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, rest_n, nn, k,
                    &alpha, B+offset+nn, HIP_R_64F, ldb, A+offset, HIP_R_64F, lda,
                    &sone, C+offset+offset*ldc+nn, HIP_R_64F, ldc, HIP_R_64F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        else
            return;
        
    }
    return;
}
void tc_ozimmu_syr2k(hipblasHandle_t handle, long int n, long int k,  double alpha, double* A, long int lda, double* B, long int ldb, double beta, double* C, long int ldc, long int nb)
{
    if(n%2||k%2) {
        double *A_, *C_, *B_;
        long int N = n, K = k, lda_, ldb_, ldc_;
        n += n%2;
        k += k%2;
        lda_ = lda + lda%2;
        ldb_ = ldb + ldb%2;
        ldc_ = ldc + ldc%2;
        hipMalloc(&A_, sizeof(double)*n*k);
        hipMalloc(&B_, sizeof(double)*n*k);
        hipMalloc(&C_, sizeof(double)*n*n);
        printf("%ld, %ld\n", n, k);
        dim3 grid1((n+31)/32, (k+31)/32);
        dim3 block(32,32);
        setInitialValueDouble<<<grid1, block>>>(n, k ,A_, lda_, 0.0);
        setInitialValueDouble<<<grid1, block>>>(n, k ,B_, ldb_, 0.0);
        dim3 grid2((n+31)/32, (n+31)/32);
        setInitialValueDouble<<<grid2, block>>>(n, n ,C_, ldc_, 1.0);
        dim3 grid3((N+31)/32, (K+31)/32);
        matrixCpyDouble<<<grid3, block>>>(N, K, A, lda, A_, lda_);
        matrixCpyDouble<<<grid3, block>>>(N, K, B, ldb, B_, ldb_);


        tc_ozimmu_syr2k_p3(handle, n, k, alpha, A_, lda_, B_, ldb_, beta, C_, ldc_, nb);
        dim3 grid4((N+31)/32, (N+31)/32);
        matrixCpyDouble<<<grid4, block>>>(N, N, C_, ldc_, C, ldc);

        printf("check ok\n");
        hipFree(A_);
        hipFree(B_);
        hipFree(C_);
    }
    else {
        tc_ozimmu_syr2k_p3(handle, n, k, alpha, A, lda, B, ldb, beta, C, ldc, nb);
    }
}