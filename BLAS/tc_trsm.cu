#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h" 

void tc_rtrsm_p2(hipblasHandle_t handle, long int m, long int n, float* A, long int lda, float* B, long int ldb, __half* hwork, long int nb)
{
    if(n <= nb)
    {
        //startTimer();
        hipblasStrsm(handle,
            HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
            HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
            m, n, &sone,
            A, lda,
            B, ldb
        );
        return;
    }
    
    tc_rtrsm_p2(handle, m, n/2, A, lda, B, ldb, hwork, nb);

    __half *Ah = hwork;
    __half *Bh = hwork+n/2*n/2;

    dim3 grid((n/2+31)/32, (n/2+31)/32);
    dim3 block(32,32);
    s2h<<<grid, block>>>(n/2, n/2, A+n/2, lda, Ah, n/2);

    dim3 grid1((m+31)/32, (n/2+31)/32);
    dim3 block1(32,32);
    s2h<<<grid1, block1>>>(m, n/2, B, ldb, Bh, m);

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n/2, n/2,
        &snegone, Bh, HIP_R_16F, m, Ah, HIP_R_16F, n/2,
        &sone, B+n/2*ldb, HIP_R_32F, ldb, HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );

    tc_rtrsm_p2(handle, m, n/2, A+n/2*lda+n/2, lda, B+n/2*ldb, ldb, hwork, nb);
}
 
void tc_trsm(hipblasHandle_t handle, long int m, long int n, float* A, long int lda, float* B, long int ldb, __half* hwork, long int nb)
{
    int length;
    int64_t* matSize = find_mat_size_trsm(n, &length);
    long int offset;
    long int rest_n = n;

    

    for(int i = length; i>=0; i--)
    {
        int64_t nn = matSize[i];
        if(i < length)
            offset += matSize[i + 1];
        else
            offset = 0;
        if(nn % 2048 == 0)
        {
            tc_rtrsm_p2(handle, m, nn, A+offset+offset*lda, lda, B+offset*ldb, ldb, hwork, nb);
        }
        else
        {
            
            hipblasStrsm(handle,
                HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
                HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
                m, nn, &sone,
                A+offset+offset*lda, lda,
                B+offset*ldb, ldb
            );
            
        }
        
    
        if(i != 0)
        {
            rest_n -=  nn;
            __half* Ah = hwork;
            
            dim3 grid((rest_n+31)/32, (nn+31)/32);
            dim3 block(32,32);
            s2h<<<grid, block>>>(rest_n, nn, A+offset+nn, lda, Ah, rest_n);
            
            __half* Bh = hwork + nn*rest_n;
            dim3 grid1((m+31)/32, (nn+31)/32);
            dim3 block1(32,32);
            s2h<<<grid1, block1>>>(m, nn, B+offset*ldb, ldb, Bh, m);

            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, rest_n, nn,
                        &snegone, Bh, HIP_R_16F, m, Ah, HIP_R_16F, rest_n,
                        &sone, B+(offset+nn)*ldb, HIP_R_32F, ldb, HIP_R_32F,
                        CUBLAS_GEMM_DEFAULT_TENSOR_OP
            );

        }
        
    


    }

}
