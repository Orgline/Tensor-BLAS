#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h" 

bool syrk_python_flag = false;
int tc_syrk_wrapper(long int n, long int k, float* A, float* C, long int nb)
{
    syrk_python_flag =true;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    __half *hwork;
    hipMalloc(&hwork, sizeof(__half)*n*k);

    float alpha = sone;
    float beta = szero;

    // float *tmp;
    // hipMalloc(&tmp, sizeof(float)*n*k);
    // dim3 grida((k+31)/32, (n+31)/32);
    // dim3 blocka(32,32);
    // transpose<<<grida, blocka>>>(k, n ,A, tmp);
    // printMatrixDeviceBlock("A.csv",n, k, A, n);
    // hipFree(tmp);

    tc_syrk(cublas_handle, n, k, alpha, A, n, beta, C, n, hwork, nb);
    dim3 gridc((n+31)/32, (n+31)/32);
    dim3 blockc(32,32);
    copy_lower_to_upper<<<gridc, blockc>>>(n, C, n);

    hipFree(hwork);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}

void tc_syrk_p2(hipblasHandle_t handle, long int n, long int k, float alpha, __half* Ah, long int lda, float beta, float* C, long int ldc, long int nb)
{
    //printf("tc_syrk_p2\n");
    hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                nb, nb, k, &alpha,
                                Ah, HIP_R_16F, lda, nb,
                                Ah, HIP_R_16F, lda, nb,
                                &beta, C, HIP_R_32F, ldc, nb+nb*lda,
                                n/nb, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    for(int i = 1;n / nb / i / 2 >= 1; i*=2)
    {
        hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                   i*nb, i*nb, k, &alpha,
                                   Ah+i*nb, HIP_R_16F, lda, 2*i*nb,
                                   Ah, HIP_R_16F, lda, 2*i*nb,
                                   &beta, C+i*nb, HIP_R_32F, ldc, 2*(i*nb+i*nb*lda),
                                   n/nb/i/2, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
}

void tc_syrk_p3(hipblasHandle_t handle, long int n, long int k,  float alpha, float* A, long int lda, float beta, float* C, long int ldc, __half* Ah, long int nb)
{
    
    int length;
    int64_t* matSize = find_mat_size_syrk(n, &length);
    int offset;
    int rest_n = n;

    if(!syrk_python_flag)
    {
        constexpr auto block_size = 256;
	    constexpr auto smem_len = block_size * 16;
	    const auto grid_size = k;
        s2h_swpipe<std::uint64_t, block_size, smem_len><<<grid_size, block_size>>>(
					n, k,
					A, lda,
					Ah, lda
					);
    }
    else
    {
        dim3 grid((k+31)/32, (n+31)/32);
        dim3 block(32,32);
        s2hTranspose<<<grid, block>>>(k, n, A, Ah);
    }

    for(int i = length; i>=0; i--)
    {

        int nn = matSize[i];
        

        if(i < length)
            offset += matSize[i + 1];
        else
            offset = 0;
        //printf("n = %ld, offset = %d\n", nn ,offset);   

        if(nn % 8192 ==0 )
        {
            tc_syrk_p2(handle, nn, k, alpha, Ah+offset, lda, beta, C+offset+offset*ldc, ldc, nb);
        }
        else
        {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nn, nn, k,
                    &alpha, Ah+offset, HIP_R_16F, lda, Ah+offset, HIP_R_16F, lda,
                    &beta, C+offset+offset*ldc, HIP_R_32F, ldc, HIP_R_32F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        if(i != 0)
        {
            rest_n -=  nn;
            //printf("rest_n = %d, nn = %d, offset = %d\n", rest_n, nn, offset);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, rest_n, nn, k,
                    &alpha, Ah+offset+nn, HIP_R_16F, lda, Ah+offset, HIP_R_16F, lda,
                    &beta, C+offset+offset*ldc+nn, HIP_R_32F, ldc, HIP_R_32F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        else
            return;
        
    }
    return;
}
void tc_syrk(hipblasHandle_t handle, long int n, long int k,  float alpha, float* A, long int lda, float beta, float* C, long int ldc, __half* Ah, long int nb)
{
    if(n%2||k%2) {
        float *A_, *C_;
        long int N = n, K = k, lda_, ldc_;
        n += n%2;
        k += k%2;
        lda_ = lda + lda%2;
        ldc_ = ldc + ldc%2;
        hipMalloc(&A_, sizeof(float)*n*k);
        hipMalloc(&C_, sizeof(float)*n*n);
        // printf("%ld, %ld\n", n, k);
        dim3 grid1((n+31)/32, (n+31)/32);
        dim3 block(32,32);
        setInitialValue<<<grid1, block>>>(n, n ,C_, ldc_, 1.0);
        dim3 grid2((n+31)/32, (k+31)/32);

        setInitialValue<<<grid2, block>>>(n, k ,A_, lda_, 0.0);
        dim3 grid3((N+31)/32, (K+31)/32);
        matrixCpy<<<grid3, block>>>(N, K, A, lda, A_, lda_);//lda lda_
        tc_syrk_p3(handle, n, k, alpha, A_, lda_, beta, C_, ldc_, Ah, nb);
        dim3 grid4((N+31)/32, (N+31)/32);
        matrixCpy<<<grid4, block>>>(N, N, C_, ldc_, C, ldc);

        printf("check ok\n");
        hipFree(A_);
        hipFree(C_);
    }
    else {
        tc_syrk_p3(handle, n, k, alpha, A, lda, beta, C, ldc, Ah, nb);
    }
    return;
}