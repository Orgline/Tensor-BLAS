#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h" 

bool syrk_python_flag = false;
int tc_syrk_wrapper(long int n, long int k, float* A, float* C, long int nb)
{
    syrk_python_flag =true;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    __half *hwork;
    hipMalloc(&hwork, sizeof(__half)*n*k);

    float alpha = sone;
    float beta = szero;

    // float *tmp;
    // hipMalloc(&tmp, sizeof(float)*n*k);
    // dim3 grida((k+31)/32, (n+31)/32);
    // dim3 blocka(32,32);
    // transpose<<<grida, blocka>>>(k, n ,A, tmp);
    // printMatrixDeviceBlock("A.csv",n, k, A, n);
    // hipFree(tmp);

    tc_syrk(cublas_handle, n, k, alpha, A, n, beta, C, n, hwork, nb);
    dim3 gridc((n+31)/32, (n+31)/32);
    dim3 blockc(32,32);
    copy_lower_to_upper<<<gridc, blockc>>>(n, C, n);

    hipFree(hwork);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}

void tc_syrk_p2(hipblasHandle_t handle, long int n, long int k, float alpha, __half* Ah, long int lda, float beta, float* C, long int ldc, long int nb)
{
    //printf("tc_syrk_p2\n");
    hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                nb, nb, k, &alpha,
                                Ah, HIP_R_16F, lda, nb,
                                Ah, HIP_R_16F, lda, nb,
                                &beta, C, HIP_R_32F, ldc, nb+nb*lda,
                                n/nb, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    for(int i = 1;n / nb / i / 2 >= 1; i*=2)
    {
        hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                   i*nb, i*nb, k, &alpha,
                                   Ah+i*nb, HIP_R_16F, lda, 2*i*nb,
                                   Ah, HIP_R_16F, lda, 2*i*nb,
                                   &beta, C+i*nb, HIP_R_32F, ldc, 2*(i*nb+i*nb*lda),
                                   n/nb/i/2, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
}

void tc_syrk(hipblasHandle_t handle, long int n, long int k,  float alpha, float* A, long int lda, float beta, float* C, long int ldc, __half* Ah, long int nb)
{
    
    int length;
    int64_t* matSize = find_mat_size_syrk(n, &length);
    int offset;
    int rest_n = n;

    if(!syrk_python_flag)
    {
        constexpr auto block_size = 256;
	    constexpr auto smem_len = block_size * 16;
	    const auto grid_size = k;
        s2h_swpipe<std::uint64_t, block_size, smem_len><<<grid_size, block_size>>>(
					n, k,
					A, lda,
					Ah, lda
					);
    }
    else
    {
        dim3 grid((k+31)/32, (n+31)/32);
        dim3 block(32,32);
        s2hTranspose<<<grid, block>>>(k, n, A, Ah);
    }

    for(int i = length; i>=0; i--)
    {

        int nn = matSize[i];
        

        if(i < length)
            offset += matSize[i + 1];
        else
            offset = 0;
        //printf("n = %ld, offset = %d\n", nn ,offset);   

        if(nn % 8192 ==0 )
        {
            tc_syrk_p2(handle, nn, k, alpha, Ah+offset, lda, beta, C+offset+offset*ldc, ldc, nb);
        }
        else
        {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nn, nn, k,
                    &alpha, Ah+offset, HIP_R_16F, lda, Ah+offset, HIP_R_16F, lda,
                    &beta, C+offset+offset*ldc, HIP_R_32F, ldc, HIP_R_32F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        if(i != 0)
        {
            rest_n -=  nn;
            //printf("rest_n = %d, nn = %d, offset = %d\n", rest_n, nn, offset);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, rest_n, nn, k,
                    &alpha, Ah+offset+nn, HIP_R_16F, lda, Ah+offset, HIP_R_16F, lda,
                    &beta, C+offset+offset*ldc+nn, HIP_R_32F, ldc, HIP_R_32F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        else
            return;
        
    }
    return;
}