#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h"

long int n, k, nb;
double alpha, beta;
bool checkFlag = false;

int parseArguments(int argc,char *argv[])
{
    if(argc < 7)
    {
        printf("Needs m, n and nb, alpha, beta, check as inputs\n");
        return -1;
    }
    n = atoi(argv[1]);
    k = atoi(argv[2]);
    nb = atoi(argv[3]);
    alpha = atof(argv[4]);
    beta= atof(argv[5]);
    if (atoi(argv[6]) == 1)
        checkFlag = true;
    else
        checkFlag = false;
    return 0;
}


__global__
void sSubstract(long int m, long int n, double* dA, long int lda, double* dB, long int ldb)
{
    long int i = (long)threadIdx.x + (long)blockDim.x*  (long)blockIdx.x;
	long int j =  (long)threadIdx.y +  (long)blockDim.y * (long)blockIdx.y;
	if (i<m && j<n) 
    {
		dA[i+j*ldb] = dA[i+j*lda] - dB[i+j*ldb];
        if(i == 40000 && j %1000 == 0)
        {    __syncthreads();
            printf("i = %d, j = %d, dA = %lf, dB=%lf\n",i, j, dA[i+j*lda], dB[i+j*ldb]);
            __syncthreads();
        }
        if(j == 40000 && i %1000 == 0)
        {    __syncthreads();
            printf("i = %d, j =%d, second dA = %lf, dB=%lf\n",i, j, dA[i+j*lda], dB[i+j*ldb]);
            __syncthreads();
        }
    }
}

int main(int argc,char *argv[])
{
    if(parseArguments(argc, argv)==-1)
        return 0;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
	// cumpsgemm::handle_t cumpsgemm_handle;
	// cumpsgemm::create(cumpsgemm_handle);

    double   *A;
    hipMalloc(&A, sizeof(double)*n*k);

    double *C;
    hipMalloc(&C, sizeof(double)*n*n);

    dim3 gridc((n+31)/32, (n+31)/32);
    dim3 blockc(32,32);


    setInitialValueDouble<<<gridc, blockc>>>(n, n ,C, n, 1.0);
    // __half *hwork;
    // hipMalloc(&hwork, sizeof(__half)*2*n*k);
    
    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, k,
                &alpha, A, HIP_R_64F, n, A, HIP_R_64F, n,
                &beta, C, HIP_R_64F, n, HIP_R_64F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    startTimer();
    // dim3 grid((n+31)/32, (k+31)/32);
    // dim3 block(32,32);
    // s2h<<<grid, block>>>(n, k, A, n, hwork, n);
    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, k,
                &alpha, A, HIP_R_64F, n, A, HIP_R_64F, n,
                &beta, C, HIP_R_64F, n, HIP_R_64F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);  

    // hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, k,
    //             &alpha, A, HIP_R_64F, n, A+n*k, HIP_R_64F, n,
    //             &beta, C, HIP_R_64F, n, HIP_R_64F,
    //             CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    float ms = stopTimer();
    
    printf("tc_gemm %dx%d takes %f ms, flops is %f\n", n, k,ms, 1.0*n*n*k/ms/1e9);

    //printf("snorm C = %f\n", snorm(n, n, C, n));
    setInitialValueDouble<<<gridc, blockc>>>(n, n ,C, n, 1.0);

    generateUniformMatrixDouble(A, n, k);
    //setInitialValue<<<gridc, blockc>>>(n, k ,A, n, 0.5);
    //printf("snorm A = %f\n", snorm(n, k, A, n));

    
    startTimer();
    tc_ozimmu_syrk(cublas_handle, n, k, alpha, A, n, beta, C, n, nb);
    ms = stopTimer();
    
    printf("tc_ozIMMU_syrk %dx%d takes %f ms, flops is %f\n", n, k,ms, 1.0*n*n*k/ms/1e9);
    
    copy_lower_to_upperDouble<<<gridc, blockc>>>(n, C, n);
    //printf("snorm C = %f\n", snorm(n, n, C, n));
    //printMatrixDeviceBlock("C.csv", n, n, C, n);
    if(checkFlag)
    {
        double *tC;
        hipMalloc(&tC, sizeof(double)*n*n);
        setInitialValueDouble<<<gridc, blockc>>>(n, n ,tC, n, 1.0);


        hipblasDsyrk(cublas_handle,
            HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
            n, k,
            &alpha,
            A, n,
            &beta,
            tC, n
        );

    
        printMatrixDeviceBlockDouble("C_p.csv", n, 1, C, n);
        copy_lower_to_upperDouble<<<gridc, blockc>>>(n, tC, n);
        //printf("snorm tC = %f\n", snorm(n, n, tC, n));
        printMatrixDeviceBlockDouble("C_a.csv", n, 1, tC, n);
        
        double sonedouble = 1.0, snegonedobule = -1.0;
        hipblasDgeam(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n,
                &sonedouble, C, n, &snegonedobule, tC, n,
                C, n);
        
        //sSubstract<<<gridc, blockc>>>(n, n, C, n, tC, n);
        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess) {
            printf("Kernel launch error: %s\n", hipGetErrorString(err));
            return 1;
        }
        
        // Synchronize the device and check for kernel execution errors
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("Kernel execution error: %s\n", hipGetErrorString(err));
            return 1;
        }
        printf("free memory is %d GB\n", free_mem()/1024/1024/1024);
        // printf("snorm C = %f\n", snorm(n, n, C, n));
        // printf("snorm tC = %f\n", snorm(n, n, tC, n));
        //hipDeviceSynchronize();
        printf("Forward error is %.6e\n",snormDouble(n, n, C, n)/snormDouble(n, n, tC, n));
        hipFree(tC);
    }
    hipFree(C);
    // hipFree(hwork);
    hipFree(A);
    
    //}

}

