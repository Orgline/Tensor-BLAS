#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h"

long int m, n, nb;
bool checkFlag = false;

int parseArguments(int argc,char *argv[])
{
    if(argc < 5)
    {
        printf("Needs m, n and nb, check as inputs\n");
        return -1;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    nb = atoi(argv[3]);
    if (atoi(argv[4]) == 1)
        checkFlag = true;
    else
        checkFlag = false;
    return 0;
}

int main(int argc,char *argv[])
{
    if(parseArguments(argc, argv)==-1)
        return 0;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
	// cumpsgemm::handle_t cumpsgemm_handle;
	// cumpsgemm::create(cumpsgemm_handle);
    double *A;
    hipMalloc(&A, sizeof(double)*n*n);
    double *B;
    hipMalloc(&B, sizeof(double)*m*n);

    // __half *hwork;
    // hipMalloc(&hwork, sizeof(__half)*(n/2*n/2+m/2*n));

    generateNormalMatrixDouble(A, n, n);
    generateUniformMatrixDouble(B, m, n);
    // dim3 gridb((m+31)/32, (n+31)/32);
    // dim3 blockb(32,32);
    // setInitialValue<<<gridb, blockb>>>(m, n ,B, m, 1.0);

    dim3 grid((n+31)/32, (n+31)/32);
    dim3 block(32,32);
    setInitialValueDouble<<<grid, block>>>(n, n ,A, n, 0.1);
    clearTriDouble<<<grid, block>>>('u', n, n, A, n);
    //printMatrixDeviceBlock("A.csv", n, n, A, n);

    double *work;
    if(checkFlag)
    {
        hipMalloc(&work, sizeof(double)*m*n);
        hipMemcpy(work, B, sizeof(double)*m*n, hipMemcpyDeviceToDevice);
    }
    
    startTimer();
    tc_ozimmu_trsm(cublas_handle, m, n, A, n, B, m, nb);
    float ms = stopTimer();

    printf("tc_ozimmu_trsm takes %f ms, flops is %f\n", ms, 1.0*m*n*n/ms/1e9);

    
    if(checkFlag)
    {
        startTimer();
        double sonedouble = 1.0, snegonedobule = -1.0;
        hipblasDtrsm(cublas_handle,
                HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
                HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
                m, n, &sonedouble,
                A, n,
                work, m
            );
        float ms = stopTimer();
        printf("Dtrsm takes %f ms, flops is %f\n", ms, 1.0*m*n*n/ms/1e9);
        // printMatrixDeviceBlock("B.csv", m, n, B, m);
        // printMatrixDeviceBlock("work.csv", m, n, work, m);

        hipblasDgeam(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n,
                &sonedouble, B, m, &snegonedobule, work, m,
                work, m);
        //printMatrixDeviceBlock("work.csv", m, n, work, m);
        printf("Forward error ||X_tc-X_cublas||/||X_cublas|| is %.6e\n", snormDouble(m,n,work,m)/snormDouble(m,n,B,m));
    }
        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess) {
            printf("Kernel launch error: %s\n", hipGetErrorString(err));
            return 1;
        }


}