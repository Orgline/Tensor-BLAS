#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h"

long int m, n, k;

int parseArguments(int argc,char *argv[])
{
    if(argc < 4)
    {
        printf("Needs m, n and k as inputs\n");
        return -1;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    k = atoi(argv[3]);
    return 0;
}

int main(int argc,char *argv[])
{
    if(parseArguments(argc, argv)==-1)
        return 0;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    float *A;
    hipMalloc(&A, sizeof(float)*m*k);

    float *B;
    hipMalloc(&B, sizeof(float)*k*n);

    float *C;
    hipMalloc(&C, sizeof(float)*m*n);

    


    //setInitialValue<<<gridc, blockc>>>(n, n ,C, n, 1.0);
    __half *hA;
    hipMalloc(&hA, sizeof(__half)*m*k);
    __half *hB;
    hipMalloc(&hB, sizeof(__half)*k*n);
     __half *hC;
    hipMalloc(&hC, sizeof(__half)*m*n);
    
    //warm up
    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k,
                &sone, hA, HIP_R_16F, m, hB, HIP_R_16F, n,
                &szero, C, HIP_R_32F, m, HIP_R_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess) {
            printf("Kernel launch error: %s\n", hipGetErrorString(err));
            return 1;
        }
        
        // Synchronize the device and check for kernel execution errors
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("Kernel execution error: %s\n", hipGetErrorString(err));
            return 1;
        }
    for(int i =0; i < 5; i++){
    
    {
        startTimer();
        dim3 gridA((m+31)/32, (k+31)/32);
        dim3 gridB((k+31)/32, (n+31)/32);
        dim3 block(32,32);
        s2h<<<gridA, block>>>(m, k, A, m, hA, m);
        s2h<<<gridB, block>>>(k, n, B, k, hB, k);
        
       hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k,
                &sone, hA, HIP_R_16F, m, hB, HIP_R_16F, n,
                &szero, C, HIP_R_32F, m, HIPBLAS_COMPUTE_32F_FAST_16F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
       float ms = stopTimer();
        
        printf("tc_gemm input: FP16, output: FP32, accumulate: FP32, %dx%dx%d takes %f ms, flops is %f\n", m, n,k, ms, 2.0*m*n*k/ms/1e9);
    }

    {
        startTimer();
        

       hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k,
                &sone, hA, HIP_R_16F, m, hB, HIP_R_16F,n,
                &szero, C, HIP_R_32F, m, HIPBLAS_COMPUTE_32F_FAST_16F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        float ms = stopTimer();
        
        printf("tc_gemm input: FP16, output: FP32, accumulate: FP32, without converting %dx%dx%d takes %f ms, flops is %f\n", m, n,k, ms, 2.0*m*n*k/ms/1e9);
    }


    {
        startTimer();

       hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k,
                &sone, A, HIP_R_32F, m, B, HIP_R_32F, n,
                &szero, C, HIP_R_32F, m, HIPBLAS_COMPUTE_32F_FAST_16F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        float ms = stopTimer();
        
        printf("tc_gemm input: FP32, output: FP32, accumulate: FP32_FAST16, %dx%dx%d takes %f ms, flops is %f\n\n\n", m, n,k, ms, 2.0*m*n*k/ms/1e9);
    }

    // {
    //     startTimer();

    //    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k,
    //             &sone, A, HIP_R_32F, m, B, HIP_R_32F, k,
    //             &szero, C, HIP_R_32F, m, HIPBLAS_COMPUTE_32F,
    //             CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    //     float ms = stopTimer();
        
    //     printf("tc_gemm input: FP32, output: FP32, accumulate: FP32, %dx%dx%d takes %f ms, flops is %f\n\n", m, n,k, ms, 2.0*m*n*k/ms/1e9);
    // }
     hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess) {
            printf("Kernel launch error: %s\n", hipGetErrorString(err));
            return 1;
        }
        
        // Synchronize the device and check for kernel execution errors
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("Kernel execution error: %s\n", hipGetErrorString(err));
            return 1;
        }
    }
    
    
    hipFree(C);
    hipFree(B);
    hipFree(hB);
    hipFree(hA);
    hipFree(A);
    
    //}

}

