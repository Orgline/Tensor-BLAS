#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h"

long int n, k, nb;
double alpha, beta;
bool checkFlag = false;

int parseArguments(int argc,char *argv[])
{
    if(argc < 7)
    {
        printf("Needs m, n and nb, alpha, beta, check as inputs\n");
        return -1;
    }
    n = atoi(argv[1]);
    k = atoi(argv[2]);
    nb = atoi(argv[3]);
    alpha = atof(argv[4]);
    beta= atof(argv[5]);
    if (atoi(argv[6]) == 1)
        checkFlag = true;
    else
        checkFlag = false;
    return 0;
}

int main(int argc,char *argv[])
{
    if(parseArguments(argc, argv)==-1)
        return 0;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
	// cumpsgemm::handle_t cumpsgemm_handle;
	// cumpsgemm::create(cumpsgemm_handle);

    double *A;
    hipMalloc(&A, sizeof(double)*n*k);

    double *B;
    hipMalloc(&B, sizeof(double)*n*k);

    double *C;
    hipMalloc(&C, sizeof(double)*n*n);

    dim3 gridc((n+31)/32, (n+31)/32);
    dim3 blockc(32,32);


    setInitialValueDouble<<<gridc, blockc>>>(n, n ,C, n, 1.0);
    // __half *hwork;
    // hipMalloc(&hwork, sizeof(__half)*n*k*2);
    
    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, k,
                &alpha, A, HIP_R_64F, n, A, HIP_R_64F, n,
                &beta, C, HIP_R_64F, n, HIP_R_64F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    startTimer();
    // dim3 grid((n+31)/32, (k+31)/32);
    // dim3 block(32,32);
    // s2h<<<grid, block>>>(n, k, A, n, hwork, n);
    // s2h<<<grid, block>>>(n, k, A, n, hwork, n);
    
    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, k,
                &alpha, A, HIP_R_64F, n, A, HIP_R_64F, n,
                &beta, C, HIP_R_64F, n, HIP_R_64F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, k,
                &alpha, A, HIP_R_64F, n, A, HIP_R_64F, n,
                &beta, C, HIP_R_64F, n, HIP_R_64F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    float ms = stopTimer();
    
    printf("Two tc_gemm %dx%d takes %f ms, flops is %f\n", n, k,ms, 2.0*n*n*k/ms/1e9);

    //printf("snorm C = %f\n", snorm(n, n, C, n));
    setInitialValueDouble<<<gridc, blockc>>>(n, n ,C, n, 1.0);

    generateUniformMatrixDouble(A, n, k);
    generateNormalMatrixDouble(B, n, k);
    //setInitialValue<<<gridc, blockc>>>(n, k ,A, n, 0.5);
    //printf("snorm A = %f\n", snorm(n, k, A, n));
    

    
    startTimer();
    tc_ozimmu_syr2k(cublas_handle, n, k, alpha, A, n, B, n, beta, C, n, nb);
    ms = stopTimer();
    
    printf("tc_ozimmu_syr2k %dx%d takes %f ms, flops is %f\n", n, k, ms, 2.0*n*n*k/ms/1e9);
    
    copy_lower_to_upperDouble<<<gridc, blockc>>>(n, C, n);
    //printf("snorm C = %f\n", snorm(n, n, C, n));
    //printMatrixDeviceBlock("C.csv", n, n, C, n);
    if(checkFlag)
    {
        double *tC;
        hipMalloc(&tC, sizeof(double)*n*n);
        setInitialValueDouble<<<gridc, blockc>>>(n, n ,tC, n, 1.0);

       

        
        startTimer();
        hipblasDsyr2k(cublas_handle,
            HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
            n, k,
            &alpha,
            A, n,
            B, n,
            &beta,
            tC, n
        );
        ms = stopTimer();
        printf("Dsyr2k %dx%d takes %f ms, flops is %f\n", n, k, ms, 2.0*n*n*k/ms/1e9);
    
        //printMatrixDeviceBlock("C_p.csv", n, n, tC, n);
        copy_lower_to_upperDouble<<<gridc, blockc>>>(n, tC, n);
        //printf("snorm tC = %f\n", snorm(n, n, tC, n));
        //printMatrixDeviceBlock("C_a.csv", n, n, tC, n);
    double sonedouble = 1.0, snegonedobule = -1.0; 

        hipblasDgeam(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n,
                &sonedouble, C, n, &snegonedobule, tC, n,
                C, n);
        
        //sSubstract<<<gridc, blockc>>>(n, n, C, n, tC, n);
        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess) {
            printf("Kernel launch error: %s\n", hipGetErrorString(err));
            return 1;
        }
        
        // Synchronize the device and check for kernel execution errors
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("Kernel execution error: %s\n", hipGetErrorString(err));
            return 1;
        }
        printf("free memory is %d GB\n", free_mem()/1024/1024/1024);
        // printf("snorm C = %f\n", snorm(n, n, C, n));
        // printf("snorm tC = %f\n", snorm(n, n, tC, n));
        //hipDeviceSynchronize();
        printf("Forward error is %.6e\n",snormDouble(n, n, C, n)/snormDouble(n, n, tC, n));
        hipFree(tC);
    }
    hipFree(C);
    // hipFree(hwork);
    hipFree(A);
    hipFree(B);
    
    //}

}

