#include "hip/hip_runtime.h"
#include "../include/TensorBLAS.h"

long int m, n, nb;
double alpha;
bool checkFlag = false;

int parseArguments(int argc,char *argv[])
{
    if(argc < 6)
    {
        printf("Needs m, n and nb, alpha, beta, check as inputs\n");
        return -1;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    nb = atoi(argv[3]);
    alpha = atof(argv[4]);
    if (atoi(argv[5]) == 1)
        checkFlag = true;
    else
        checkFlag = false;
    return 0;
}

int main(int argc,char *argv[])
{
    if(parseArguments(argc, argv)==-1)
        return 0;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
	// cumpsgemm::handle_t cumpsgemm_handle;
	// cumpsgemm::create(cumpsgemm_handle);

    double *A;
    hipMalloc(&A, sizeof(double)*m*m);

    double *B;
    hipMalloc(&B, sizeof(double)*m*n);

    double *C;
    hipMalloc(&C, sizeof(double)*m*n);

    generateUniformMatrixDouble(A, m, m);

    generateUniformMatrixDouble(B, m, n);

    dim3 gridc((m+31)/32, (n+31)/32);
    dim3 blockc(32,32);

    setInitialValueDouble<<<gridc, blockc>>>(m, n ,C, m, 0.0);

    dim3 grida((m+31)/32, (m+31)/32);
    clearTriDouble<<<grida, blockc>>>('u', m, m, A, m);

    // __half *hwork;
    // hipMalloc(&hwork, sizeof(__half)*(m*n+m*m));
    
    // hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, k,
    //             &alpha, hwork, HIP_R_16F, n, hwork, HIP_R_16F, n,
    //             &beta, C, HIP_R_32F, n, HIP_R_32F,
    //             CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, m,
                &alpha, A, HIP_R_64F, m, A, HIP_R_64F, m,
                &szero, C, HIP_R_64F, m, HIP_R_64F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    startTimer();
    
    // s2h<<<grida, blockc>>>(m, m, A, m, hwork, m);
    // s2h<<<gridc, blockc>>>(m, n, B, m, hwork, m);
    

    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, m,
                &alpha, A, HIP_R_64F, m, A, HIP_R_64F, m,
                &szero, C, HIP_R_64F, m, HIP_R_64F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    float ms = stopTimer();
    
    printf("tc_64gemm %dx%d takes %f ms, flops is %f\n", m, n, ms, 1.0*m*n*m/ms/1e9);

    //printf("snorm C = %f\n", snorm(n, n, C, n));
    setInitialValueDouble<<<gridc, blockc>>>(m, n ,C, m, 0.0);

    
    //setInitialValue<<<gridc, blockc>>>(n, k ,A, n, 0.5);
    //printf("snorm A = %f\n", snorm(n, k, A, n));

    
    startTimer();
    tc_ozimmu_trmm(cublas_handle, m, n, alpha, A, m, B, m, C, m, nb);
    ms = stopTimer();
    //printMatrixDeviceBlock("tC.csv", m, n, C, m);
    
    printf("tc_ozimmu_trmm %dx%d takes %f ms, flops is %f\n", m, n, ms, 1.0*m*n*m/ms/1e9);
    
    if(checkFlag)
    {
        double *tC;
        hipMalloc(&tC, sizeof(double)*m*n);
        setInitialValueDouble<<<gridc, blockc>>>(m, n ,tC, m, 0.0);
        startTimer();

        hipblasDtrmm(cublas_handle,
                    HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
                    HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                    m, n, &alpha,
                    A, m, B, m, tC, m);
        ms = stopTimer();
    //printMatrixDeviceBlock("tC.csv", m, n, C, m);
    
        printf("tc_cublas_Dtrmm %dx%d takes %f ms, flops is %f\n", m, n, ms, 1.0*m*n*m/ms/1e9);
        // printMatrixDeviceBlock("A.csv", m, m, A, m);
        // printMatrixDeviceBlock("B.csv", m, n, B, m);
        // printMatrixDeviceBlock("C.csv", m, n, tC, m);
        //printf("snorm tC = %f\n", snorm(n, n, tC, n));
        //printMatrixDeviceBlock("C_a.csv", n, n, tC, n);
        
        double sonedouble = 1.0, snegonedobule = -1.0;
        hipblasDgeam(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n,
                &sonedouble, C, m, &snegonedobule, tC, m,
                C, m);
        
        //sSubstract<<<gridc, blockc>>>(n, n, C, n, tC, n);
        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess) {
            printf("Kernel launch error: %s\n", hipGetErrorString(err));
            return 1;
        }
        
        // Synchronize the device and check for kernel execution errors
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("Kernel execution error: %s\n", hipGetErrorString(err));
            return 1;
        }
        printf("free memory is %d GB\n", free_mem()/1024/1024/1024);
        printf("snorm C = %f\n", snormDouble(m, n, C, m));
        printf("snorm tC = %f\n", snormDouble(m, n, tC, m));
        //hipDeviceSynchronize();
        printf("Forward error is %.6e\n",snormDouble(m, n, C, m)/snormDouble(m, n, tC, m));
        hipFree(tC);
    }
    hipFree(C);
    // hipFree(hwork);
    hipFree(A);
    hipFree(B);
    
    //}

}

